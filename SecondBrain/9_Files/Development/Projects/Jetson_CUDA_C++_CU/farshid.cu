// #include <iostream>
// #include <hip/hip_runtime.h>
// #include <opencv2/opencv.hpp>
// #include <opencv2/highgui.hpp>
// #include <opencv2/dnn.hpp>

// using namespace std;
// using namespace cv;
// using namespace cv::cuda;

// int main() {
//     Mat src1(640, 480, CV_8UC4, Scalar::all(0));
//     rectangle(src1, Rect(50, 50, 300, 200), Scalar(0, 0, 255, 128), 30);    
//     GpuMat d_src1;
//     d_src1.upload(src1);    
//     Mat result;
//     d_src1.download(result);    
//     imshow("Result Image", result);
//     waitKey(0);
//     return 0;
// }


// #include <iostream>
// #include <hip/hip_runtime.h>
// #include <opencv2/opencv.hpp>
// #include <opencv2/highgui.hpp>
// #include <opencv2/dnn.hpp>

// using namespace std;
// using namespace cv;
// using namespace cv::cuda;

// int main() {
//     Mat src1(640, 480, CV_8UC4, Scalar::all(0));
//     rectangle(src1, Rect(50, 50, 300, 200), Scalar(0, 0, 255, 128), 30);
//     GpuMat d_src1;
//     d_src1.upload(src1);
//     vector<Rect> bboxes;
//     vector<float> scores;
//     vector<int> result;
//     // TODO: populate bboxes and scores with bounding box data
//     dnn::NMSBoxes(bboxes, scores, 0.4, 0.6, result);
//     GpuMat d_result;
//     d_result.upload(result);
//     Mat result_cpu;
//     d_result.download(result_cpu);
//     imshow("Result Image", src1);
//     waitKey(0);
//     return 0;
// }


#include <iostream>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/dnn.hpp>
#include <opencv2/cudaobjdetect.hpp>

using namespace std;
using namespace cv;
using namespace cv::cuda;

int main() {
    // Load the face and eye cascade classifiers
    cv::CascadeClassifier face_cascade;
    cv::CascadeClassifier eyes_cascade;

    if (!face_cascade.load("/home/tiziran/code/image_web/tiziran/lib/python3.8/site-packages/cv2/data/haarcascade_frontalface_default.xml")) {
        cout << "Error loading face cascade\n";
        return -1;
    }
    if (!eyes_cascade.load("/home/tiziran/code/image_web/tiziran/lib/python3.8/site-packages/cv2/data/haarcascade_eye.xml")) {
        cout << "Error loading eyes cascade\n";
        return -1;
    }

    // Open the default camera
    VideoCapture cap(0);
    if (!cap.isOpened()) {
        cout << "Error opening video capture\n";
        return -1;
    }

    Mat frame, gray_frame;
    GpuMat d_frame, d_gray_frame;
     while (cap.read(frame)) {
        // Convert frame to grayscale
        cvtColor(frame, gray_frame, COLOR_BGR2GRAY);
        d_frame.upload(frame);
        d_gray_frame.upload(gray_frame);

        // Detect faces
        std::vector<Rect> faces;
        face_cascade.detectMultiScale(gray_frame, faces);

        for (size_t i = 0; i < faces.size(); i++) {
            Point center(faces[i].x + faces[i].width / 2, faces[i].y + faces[i].height / 2);
            ellipse(frame, center, Size(faces[i].width / 2, faces[i].height / 2), 0, 0, 360, Scalar(255, 0, 255), 4);

            Mat faceROI = gray_frame(faces[i]);
            std::vector<Rect> eyes;
            eyes_cascade.detectMultiScale(faceROI, eyes);

            for (size_t j = 0; j < eyes.size(); j++) {
                Point eye_center(faces[i].x + eyes[j].x + eyes[j].width / 2, faces[i].y + eyes[j].y + eyes[j].height / 2);
                int radius = cvRound((eyes[j].width + eyes[j].height) * 0.25);
                circle(frame, eye_center, radius, Scalar(255, 0, 0), 4);
            }
        }

        // Save the frame with detections
        imwrite("output.jpg", frame);

        // Break the loop after saving one frame (for testing purposes)
        break;
    }

    return 0;
}
