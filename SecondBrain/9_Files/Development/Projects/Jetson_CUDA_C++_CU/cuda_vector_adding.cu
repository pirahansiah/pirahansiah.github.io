#include "hip/hip_runtime.h"
/*
export PATH=/usr/local/cuda/bin${PATH:+:${PATH}}
export LD_LIBRARY_PATH=/usr/local/cuda/lib64${LD_LIBRARY_PATH:+:${LD_LIBRARY_PATH}}
//nvcc -std=c++11 -o cuda1 cuda_vector_adding.cu $(pkg-config --cflags --libs opencv4)
nvcc -std=c++11 -o cuda1 cuda_vector_adding.cu $(pkg-config --cflags --libs opencv4) -Xcudafe "--diag_suppress=overloaded_virtual"
*/
#include <iostream>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <hip/hip_runtime.h>
#include <opencv2/core/opengl.hpp>
#include <opencv2/highgui.hpp>
//#include <opencv2/cudaimgproc.hpp>

using namespace std;
using namespace cv;
using namespace cv::cuda;

__global__ void matrixMultiply(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (row < N && col < N) {
        for (int i = 0; i < N; i++) {
            sum += A[row * N + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}
int main() {
    std::cout << cv::getBuildInformation() << std::endl;
    cout << "This program demonstrates using alphaComp" << endl;
    cout << "Press SPACE to change compositing operation" << endl;
    cout << "Press ESC to exit" << endl;

    namedWindow("First Image", WINDOW_NORMAL);
    namedWindow("Second Image", WINDOW_NORMAL);
    namedWindow("Result", WINDOW_OPENGL);

    setGlDevice();

    Mat src1(640, 480, CV_8UC4, Scalar::all(0));
    Mat src2(640, 480, CV_8UC4, Scalar::all(0));

    rectangle(src1, Rect(50, 50, 200, 200), Scalar(0, 0, 255, 128), 30);
    rectangle(src2, Rect(100, 100, 200, 200), Scalar(255, 0, 0, 128), 30);

    GpuMat d_src1(src1);
    GpuMat d_src2(src2);

    GpuMat d_res;

    imshow("First Image", src1);
    imshow("Second Image", src2);
    std::cout << cv::getBuildInformation() << std::endl;
    cv::waitKey(0);
/*
    int alpha_op = ALPHA_OVER;

    const char* op_names[] =
    {
        "ALPHA_OVER", "ALPHA_IN", "ALPHA_OUT", "ALPHA_ATOP", "ALPHA_XOR", "ALPHA_PLUS", "ALPHA_OVER_PREMUL", "ALPHA_IN_PREMUL", "ALPHA_OUT_PREMUL",
        "ALPHA_ATOP_PREMUL", "ALPHA_XOR_PREMUL", "ALPHA_PLUS_PREMUL", "ALPHA_PREMUL"
    };
/*
    for(;;)
    {
        cout << op_names[alpha_op] << endl;

        alphaComp(d_src1, d_src2, d_res, alpha_op);

        imshow("Result", d_res);

        char key = static_cast<char>(waitKey());

        if (key == 27)
            break;

        if (key == 32)
        {
            ++alpha_op;

            if (alpha_op > ALPHA_PREMUL)
                alpha_op = ALPHA_OVER;
        }
    }
*/
    /////
    cv::Mat image = cv::Mat::zeros(256, 256, CV_8UC3);
    image.setTo(cv::Scalar(255, 0, 0));  // Blue color

    // Create a window for display.
    cv::namedWindow("Display window", cv::WINDOW_AUTOSIZE);

    // Show our image inside it.
    cv::imshow("Display window", image);

    // Wait for a keystroke in the window
    cv::waitKey(0);




    std::string videoPath="/home/tiziran/farshid/depthai-python/examples/LLM.mp4";
    

    // Create a VideoCapture object
    cv::VideoCapture cap(videoPath);

    // Check if video opened successfully
    if (!cap.isOpened()) {
        std::cerr << "Error: Could not open video." << std::endl;
        return -1;
    }

    // Get various properties of the video
    double width = cap.get(cv::CAP_PROP_FRAME_WIDTH);
    double height = cap.get(cv::CAP_PROP_FRAME_HEIGHT);
    double fps = cap.get(cv::CAP_PROP_FPS);
    double totalFrames = cap.get(cv::CAP_PROP_FRAME_COUNT);
    int codecCode = static_cast<int>(cap.get(cv::CAP_PROP_FOURCC));
    char codecChars[5] = {0};
    codecChars[0] = codecCode & 0xFF;
    codecChars[1] = (codecCode >> 8) & 0xFF;
    codecChars[2] = (codecCode >> 16) & 0xFF;
    codecChars[3] = (codecCode >> 24) & 0xFF;
    std::string codec(codecChars);

    // Print the details
    std::cout << "Video Width: " << width << std::endl;
    std::cout << "Video Height: " << height << std::endl;
    std::cout << "Frame Rate (FPS): " << fps << std::endl;
    std::cout << "Total Number of Frames: " << totalFrames << std::endl;
    std::cout << "Codec: " << codec << std::endl;

    // Release the video capture object
    cap.release();


    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device Number: " << i << std::endl;
        std::cout << "  Device name: " << prop.name << std::endl;
        std::cout << "  Memory Clock Rate (KHz): " << prop.memoryClockRate << std::endl;
        std::cout << "  Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
        std::cout << "  Peak Memory Bandwidth (GB/s): "
                  << 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6 << std::endl;
    }
    return 0;
}